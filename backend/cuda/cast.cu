
#include <hip/hip_runtime.h>



template <int nt, int vt, typename T1, typename T2>
__global__ void Cast(T1* input, T2* output, int numel) {
  int tid = threadIdx.x;
  int idx = nt * vt * blockIdx.x + tid;

  for (int v = 0; v < vt; ++v) {
    if (idx >= numel) return;

    output[idx] = scalar_cast<T2>(input[idx]);

    idx += nt;
  }
}


