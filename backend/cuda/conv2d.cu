#include "hip/hip_runtime.h"

#include "threads.cuh"
#include "datatype_dispatch.h"

namespace SimpleNet {

// N = (w - size + 2p) / stride + 1
template <int nt, int vt, typename T>
__global__ void Conv2d(T* input, T* kernel, int i_h, int i_w,
                       T* output, int o_h, int o_w,
                       int size, int stride, int pad, int o_numel) {
  int tid = threadIdx.x;
  int idx = nt * vt * blockIdx.x + tid;

  for (int v = 0; v < vt; ++v) {
    if (idx >= o_numel) return;

    int o_row = idx / o_w;
    int o_col = idx - row * o_w;

    int i_start_row = o_row * stride;
    int i_start_col = o_col * stride;

    T sum = static_cast<T>(0);
    for (int i = 0; i < size; ++i) {
      for (int j = 0; j < size; ++j) {
        int i_row = i_start_row + i;
        int i_col = i_start_col + j;
        // padding area
        if (i_row < pad || i_row > (i_h + p) || i_col < pad || i_col > (i_w + p)) {
          return;
        }
        int i_idx = (i_start_row + i) * o_w + j + i_start_col;
        int k_idx = i * size + j;
        sum += input[i_idx] * kernel[k_idx];
      }
    }

    output[idx] = sum;

    idx += nt;
  }
}

void LaunchConv2d(void* input, void* kernel, int i_h, int i_w,
                  void* output, int o_h, int o_w,
                  int size, int stride, int pad,
                  SNDataType_t data_type) {
  int o_numel = o_h * o_w;

  dim3 block(threads_per_block);
  dim3 grid((o_numel + BlockWorkSize() - 1) / BlockWorkSize());

  DISPATH_TYPE_FLOAT(data_type, [=]() {
    Conv2d<threads_per_block, thread_work_size, scalar_t><<<grid, block, 0, 0>>>(static_cast<scalar_t *>(input),
                                                                                 static_cast<scalar_t *>(kernel),
                                                                                 i_h, i_w,
                                                                                 static_cast<scalar_t *>(output),
                                                                                 o_h, o_w,
                                                                                 size, stride, pad, o_numel);
  });
};

}
